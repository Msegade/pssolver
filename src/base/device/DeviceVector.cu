#include "DeviceVector.hpp"
#include "../host/HostVector.hpp"

#include "cuda_kernels_vector.hpp"
#include "cuda_utils.h"
#include "cuda_variables.h"

#include <cassert>
#include <cstring>
#include <iostream>
#include <cmath>

namespace pssolver
{

template <typename ValueType>
DeviceVector<ValueType>::DeviceVector()
{
        
}

template <typename ValueType>
DeviceVector<ValueType>::~DeviceVector()
{
    checkCudaErrors(hipFree(d_mData)); 
}

template <typename ValueType>
void DeviceVector<ValueType>::Allocate (const int size)
{
    assert(size > 0);
    mSize = size;
    checkCudaErrors(hipMalloc(&d_mData, mSize*sizeof(double)));
}

template <typename ValueType>
void DeviceVector<ValueType>::SetVal (const ValueType val)
{
    dim3 BlockSize(BLOCKSIZE);
    dim3 GridSize( mSize / BLOCKSIZE +1);
    kernel_vector_fill <<<GridSize, BlockSize>>>(mSize, d_mData, val);
}

template <typename ValueType>
void DeviceVector<ValueType>::CopyFrom(const BaseVector<ValueType>& src)
{
    const HostVector<ValueType> *cast_vec; 
    cast_vec = dynamic_cast<const HostVector<ValueType>*> (&src);

    checkCudaErrors(hipMemcpy(d_mData, cast_vec->mData, mSize*sizeof(double),
                    hipMemcpyHostToDevice));

}

template <typename ValueType>
void DeviceVector<ValueType>::Add(
                        const BaseVector<ValueType> &otherVector)
{
    const DeviceVector<ValueType> *cast_vec = 
        dynamic_cast<const DeviceVector<ValueType>*> (&otherVector);

    dim3 BlockSize(BLOCKSIZE);
    dim3 GridSize( mSize / BLOCKSIZE +1);
    kernel_vector_add <<<GridSize, BlockSize>>>
                                    (mSize, d_mData, cast_vec->d_mData);

}

template <typename ValueType>
void DeviceVector<ValueType>::Add(
                        const BaseVector<ValueType> &v1,
                        const BaseVector<ValueType> &v2)

{
    const DeviceVector<ValueType> *cast_v1 = 
        dynamic_cast<const DeviceVector<ValueType>*> (&v1);
    const DeviceVector<ValueType> *cast_v2 = 
        dynamic_cast<const DeviceVector<ValueType>*> (&v2);

    dim3 BlockSize(BLOCKSIZE);
    dim3 GridSize( mSize / BLOCKSIZE +1);
    kernel_vector_add <<<GridSize, BlockSize>>>
                                    (mSize, d_mData,
                                     cast_v1->d_mData,
                                     cast_v2->d_mData);

}

template <typename ValueType>
double DeviceVector<ValueType>::Norm(void) const
{
    double result = 0.0;
    return result;

}

template class DeviceVector<double>;
template class DeviceVector<float>;
template class DeviceVector<int>;

}
