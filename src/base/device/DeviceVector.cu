#include "DeviceVector.hpp"
#include "../host/HostVector.hpp"

#include "cuda_kernels_vector.hpp"
#include "cuda_utils.h"
#include "cuda_variables.h"

#include <cassert>
#include <cstring>
#include <iostream>
#include <cmath>

namespace pssolver
{

template <typename ValueType>
DeviceVector<ValueType>::DeviceVector()
{
        
}

template <typename ValueType>
DeviceVector<ValueType>::~DeviceVector()
{
    checkCudaErrors(hipFree(d_mData)); 
}

template <typename ValueType>
void DeviceVector<ValueType>::Allocate (const int size)
{
    assert(size > 0);
    mSize = size;
    checkCudaErrors(hipMalloc(&d_mData, mSize*sizeof(double)));
}

template <typename ValueType>
void DeviceVector<ValueType>::SetVal (const ValueType val)
{
    dim3 BlockSize(BLOCKSIZE);
    dim3 GridSize( mSize / BLOCKSIZE +1);
    kernel_fill_vector <<<GridSize, BlockSize>>>(mSize, d_mData, val);
}

template <typename ValueType>
void DeviceVector<ValueType>::CopyFrom(const BaseVector<ValueType>& src)
{
    const HostVector<ValueType> *cast_vec; 
    cast_vec = dynamic_cast<const HostVector<ValueType>*> (&src);

    checkCudaErrors(hipMemcpy(d_mData, cast_vec->mData, mSize*sizeof(double),
                    hipMemcpyHostToDevice));

}

template class DeviceVector<double>;
template class DeviceVector<float>;
template class DeviceVector<int>;

}
